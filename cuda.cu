#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

__device__
unsigned long isPrime(unsigned long n)
{
  // Corner case
  if (n <= 1)
    return 0;

  // Check from 2 to n-1
  for (int i = 2; i <= sqrtf(static_cast<float>(n)); i++)
    if (n % i == 0)
      return 0;

  return 1;
}

__global__
void CheckHowManyPrimes(unsigned long numbers[], unsigned long long *primes, long size)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
  {
    if (isPrime(numbers[index]))
    {
      atomicAdd(primes, 1);
    }
  }
}

int main(int argc, char **argv)
{

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);

  // program input argument
  long inputArgument = ins__args.arg;
  unsigned long int *numbers = (unsigned long int *)malloc(inputArgument * sizeof(unsigned long int));
  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);

  // run your CUDA kernel(s) here
  unsigned long *d_numbers;
  unsigned long long *d_primes;
  unsigned long long primes = 0;

  hipMalloc(&d_numbers, inputArgument * sizeof(unsigned long));
  hipMalloc(&d_primes, sizeof(unsigned long long));

  hipMemcpy(d_numbers, numbers, inputArgument * sizeof(unsigned long), hipMemcpyHostToDevice);
  hipMemcpy(d_primes, &primes, sizeof(unsigned long long), hipMemcpyHostToDevice);

  int blockSize = 256;
  int gridSize = (inputArgument + blockSize - 1) / blockSize;

  CheckHowManyPrimes<<<gridSize, blockSize>>>(d_numbers, d_primes, inputArgument);

  hipMemcpy(&primes, d_primes, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  printf("Number of primes: %lld\n", primes);

  hipFree(d_numbers);
  hipFree(d_primes);

  free(numbers);

  // synchronize/finalize your CUDA computations

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);
}
