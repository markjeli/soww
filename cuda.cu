#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "numgen.c"

int main(int argc, char **argv)
{

  Args ins__args;
  parseArgs(&ins__args, &argc, argv);

  // program input argument
  long inputArgument = ins__args.arg;
  unsigned long int *numbers = (unsigned long int *)malloc(inputArgument * sizeof(unsigned long int));
  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);

  // run your CUDA kernel(s) here

  for (long i = 0; i < inputArgument; i++)
  {
    printf("%ld\n", numbers[i]);
  }

  // synchronize/finalize your CUDA computations

  gettimeofday(&ins__tstop, NULL);
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);
}
